#include "hip/hip_runtime.h"
#include <stdio.h>

int main(void) {
	hipDeviceProp_t prop;
	int count;

	hipGetDeviceCount(&count);
	for (int i=0; i<count; i++){
		hipGetDeviceProperties(&prop, i);

		printf("---General information for device %d---\n", i);
		printf("Name : %s\n", prop.name);
		printf("Compute Capability : %d.%d\n", prop.major, prop.minor);

		printf("\n---Memory information for device %d---\n", i);
		printf("Total global Mem : %lu\n", prop.totalGlobalMem);
		printf("Total constant Mem : %lu\n", prop.totalConstMem);
	}

	return 0;
}